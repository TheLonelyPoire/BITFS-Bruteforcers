#include "hip/hip_runtime.h"
#include <fstream>
#include <cstring>
#include <string> 
#include <unordered_map>
#include <iostream>
#include <chrono>
#include <cmath>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include "../Common/BloomFilter.cuh"
#include "../Common/IOUtils.hpp"
#include "../Common/Trig.cuh"
#include "../Common/VMath.cuh"

#include "BullyImpactStructs.hpp"

# define MAX_INITIALS_PER_ARRIVAL 5000000
# define MAX_UNIQUE 10000
# define MAX_GOOD_IMPACTS_PER_ARRIVAL 500000
# define MAX_IMPACTS 1000000

using namespace BITFS;


// so, the masterFilter is going to be the bloom filter. initialImpactsLog is going to be where we store the immediate post-impact
// state of a bully if it looks good and isn't a duplicate. goodImpactsLog is going to be where we store how long the bully is stable 
// for. finalImpactsLog is going to be where we accumulate all our solutions in. nInitials and nImpacts are incremented to tell us
// how far in the impact logs to look for, and nSolutions is incremented to tell us how big the final impact log is going to be.
__device__ int* scratchpadAlpha;
__device__ int* scratchpadBeta;
__device__ int* scratchpadGamma;
__device__ int nUnchecked;
__device__ int nChecked;
__device__ BullyData* initialImpactsLog;
__device__ int nInitials;
__device__ SecondaryData* goodImpactsLog;
__device__ int nImpacts;
__device__ ImpactData* finalImpactsLog;
__device__ int nSolutions = 0;




// The bully collision simulator.
__device__ BullyData sim_bully_collision(float* marioPos, float* bullyPos, int facingAngle, int bullyMovingAngle, float marioVel, float bullySpeed) {

    float offsetX = marioPos[0] - bullyPos[0];
    float offsetZ = marioPos[2] - bullyPos[2];

    // Removed unecessary distance calculation with sqrtf

    int pushAngle;

    if (offsetX * offsetX + offsetZ * offsetZ == 0.0f) {
        pushAngle = fix(facingAngle);
    }
    else {
        pushAngle = fix(atan2s(offsetZ, offsetX));
    }

    int bullyOldYaw = fix(bullyMovingAngle);

    float newMarioX = bullyPos[0] + 115.0f * sm64_sins(pushAngle);
    float newMarioZ = bullyPos[2] + 115.0f * sm64_coss(pushAngle);

    float marioSpeed = -1.0f * marioVel;
    int marioYaw = fix(facingAngle + 0x8000);

    float marioVelX = marioSpeed * sm64_sins(marioYaw);
    float marioVelZ = marioSpeed * sm64_coss(marioYaw);

    float rx = bullyPos[0] - newMarioX;
    float rz = bullyPos[2] - newMarioZ;

    float bullyVelX = bullySpeed * sm64_sins(bullyOldYaw);
    float bullyVelZ = bullySpeed * sm64_coss(bullyOldYaw);

    float projectedV1 = (rx * marioVelX + rz * marioVelZ) / (rx * rx + rz * rz);
    float projectedV2 = (-rx * bullyVelX - rz * bullyVelZ) / (rx * rx + rz * rz);

    bullyVelX += (53.0f / 73.0f) * projectedV1 * rx - projectedV2 * -rx;
    bullyVelZ += (53.0f / 73.0f) * projectedV1 * rz - projectedV2 * -rz;

    int bullyYaw = fix(atan2s(bullyVelZ, bullyVelX));
    float bullyVel = sqrtf(bullyVelX * bullyVelX + bullyVelZ * bullyVelZ);

    struct BullyData solution;
    solution.posBully[0] = bullyPos[0];
    solution.posBully[1] = bullyPos[1];
    solution.posBully[2] = bullyPos[2];
    solution.angle = bullyYaw;
    solution.velBully = bullyVel;

    return solution;

}



// This takes a bully and time-evolves it for up to "ticks" ticks of the HAU-clock. If the bully "survives" (read: stays in the FST
// position when it's in the main universe, and doesn't zip off to a PU) for x ticks of the HAU-clock then this function spits out 
// x * 16 as an estimate for the number of frames the bully is stable for.
__device__ int stability_frames(BullyData bully, int ticks) {

    
    // we aren't keeping track of y elevation, so we have a bunch of floats.
    // location is where the bully starts off, in the main universe.
    // nearoob is where the bully tries (and fails) to move to, out-of-bounds.
    // farlocation is where the bully reflects to, somewhere off in the PU's.
    // faroob is where the bully tries (and fails) to move to after that, somewhere very far away
    // and then hopefully the bully reflects back to its starting location.
    float location[2];
    float nearoob[2];
    float farlocation[2];
    float faroob[2];

    // initialize the bully starting angle and position. It'll evolve as time goes on.
    int theta = fix(bully.angle);
    location[0] = bully.posBully[0];
    location[1] = bully.posBully[2];


    // iterate over intervals of 16-frames (so each tick of this for loop simulates a forward-then-back pair of bully motions).
    // after each tick, we can skip forward 14 frames because that will land us in the middle of the bully performing its next
    // meaningfully distinct behavior.
    for (int i = 0; i < ticks; i++) {

        
        // populate the positions. First nearoob, then the angle flips around to send the bully to farlocation, then faroob
        // then the angle flips around to send the bully to location again.
        nearoob[0] = location[0] + sm64_sins(theta) * bully.velBully;
        nearoob[1] = location[1] + sm64_coss(theta) * bully.velBully;
        theta = fix(theta + 32767);
        farlocation[0] = location[0] + sm64_sins(theta) * bully.velBully;
        farlocation[1] = location[1] + sm64_coss(theta) * bully.velBully;
        faroob[0] = farlocation[0] + sm64_sins(theta) * bully.velBully;
        faroob[1] = farlocation[1] + sm64_coss(theta) * bully.velBully;
        theta = fix(theta + 32767);
        location[0] = farlocation[0] + sm64_sins(theta) * bully.velBully;
        location[1] = farlocation[1] + sm64_coss(theta) * bully.velBully;

        
        // now, as stated, there are three critical checks we need to do. First, ensuring that nearoob and faroob are both
        // out of bounds. Second, ensuring that the new location (where the bully returned to when it reflected) is at the FST 
        // coordinates. an important note is that i is actually "how many "ticks" of the bully clock have passed" and it must be 
        // multiplied by 16 to estimate the number of frames, since one "tick" happens every 16 frames.
        if ( (short)(int)nearoob[0] > -8192 && (short)(int)nearoob[0] < 8192 && (short)(int)nearoob[1] > -8192 && (short)(int)nearoob[1] < 8192) {
            return i * 16;
        }
        if ( (short)(int)faroob[0] > -8192 && (short)(int)faroob[0] < 8192 && (short)(int)faroob[1] > -8192 && (short)(int)faroob[1] < 8192) {
            return i * 16;
        }
        if (fabs(location[0] + 3120.0f) > 0.1f || fabs(location[1] + 896.0f) > 0.1f) {
            return i * 16;
        }

            
        // one last bit, since i is bully clock "ticks" and each tick is 16 frames and we've advanced 2 frames while simulating
        // thereby making theta 2 less than what it was, we need to subtract 14 from theta so that, on the next "tick", 
        // theta starts off 16 frames ahead of where it was on the previous tick, and the angle is 16 less than what it started as.
        theta = fix(theta - 14);
    }


    // if we got out of the loop, the bully is stable for a minimum of "ticks" ticks of the HAU-clock. return this fact.
    return ticks * 16;
}



__global__ void copy_pointers_to_gpu(BullyData* p1, SecondaryData* p2, ImpactData* p3, int* p4, int* p5, int* p6) {
    initialImpactsLog = p1;
    goodImpactsLog = p2;
    finalImpactsLog = p3;
    scratchpadAlpha = p4;
    scratchpadBeta = p5;
    scratchpadGamma = p6;
}



// this gets a bully collision, checks whether a mario-bully collision is possible, and if so, simulates the impact.
// then impacts with the wrong bully parity or which make the bully go too slow or too fast are thrown out.
// a bloom filter for unique impacts is run, and only after that is the collision saved. We have up to 4 million things to test
// so it's useful to have less data get to the end to run through the more expensive simulation step later.
// a critical warning about this, however. This means that for each solution that actually gets written to the end file
// there are multiple bully positions which would produce that same impact! So I should probably come up with some code later
// to take a promising solution and find the bully positions that produce that exact solution.
__global__ void initial_assessment(ApproachData mario, BullyData bullyCentral, int nx, int nz, float minx, float minz, float gran) {


    // get the number of increments in the x and z direction from the thread id. The % nz is intended.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nx * nz) {
        return;
    }
    int overx = idx % nx;
    int overz = (idx - overx) / nx;

    // load up our bully data.
    struct BullyData bully;
    bully.posBully[0] = minx + (overx * gran);
    bully.posBully[1] = bullyCentral.posBully[1];
    bully.posBully[2] = minz + (overz * gran);
    bully.angle = bullyCentral.angle;
    bully.velBully = bullyCentral.velBully;


    // compute the distance that Mario is at from the bully position. If it's too high, automatically give up.
    if (find_dis(mario.posArrive, bully.posBully) > 63.0f) {
        return;
    }


    // ok, Mario can impact with the bully. Simulate the impact, to update the bully state.
    bully = sim_bully_collision(mario.posArrive, bully.posBully, mario.facingArrive, bully.angle, mario.velArrive, bully.velBully);


    // throw out if the angle impact parity is odd (because that moves the pivot) or the bully velocity is too low (<400 million)
    // or too high (>1 billion).
    if (bully.angle % 2 == 1 || bully.velBully < 4.0e+08 || bully.velBully > 1.0e+09) {
        return;
    }


    // see if the bully returns to its FST location. stability_frames(bully, 1) would be 16 if 
    // the bully returns to the FST location and 0 otherwise.
    bool toFST = (stability_frames(bully, 1) > 0);
    if (!toFST) {
        return;
    }


    // increment our solution counter and record our solution.
    int solIdx = atomicAdd(&nInitials, 1);
    if (solIdx > MAX_INITIALS_PER_ARRIVAL) {
        return;
    }
    struct BullyData* data = &(initialImpactsLog[solIdx]);
    data->posBully[0] = bully.posBully[0];
    data->posBully[1] = bully.posBully[1];
    data->posBully[2] = bully.posBully[2];
    data->angle = bully.angle;
    data->velBully = bully.velBully;
}




// evolves the bullies in time to see how long they last.
__global__ void time_evolution(int size) {


    // get the place-to-look-at from the thread id.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) {
        return;
    }

    
    // simulate the bully time evolution and throw it out if the bully is sufficiently unstable (< 400 frames ie about 15 seconds)
    int duration = stability_frames(initialImpactsLog[scratchpadGamma[idx]], 200);
    if (duration < 400) {
        return;
    }


    // increment the solution counter and record our pointer to where we have to look, and how stable it is.
    int solIdx = atomicAdd(&nImpacts, 1);
    if (solIdx > MAX_GOOD_IMPACTS_PER_ARRIVAL) {
        return;
    }
    struct SecondaryData* data = &(goodImpactsLog[solIdx]);
    data->tag = idx;
    data->frames = duration;
}




// adds the data to our ever-growing solution list. identifier tells us which mario-arrival to link the bully data up with.
__global__ void append_info(ApproachData approach, int size) {

    // initialize our counter
    int counter = 0;


    // iterate up to the number of impacts we're looking at.
    for (int i = 0; i < size; i++) {


        // continue if we are over our bound.
        if (nSolutions + counter >= MAX_IMPACTS) {
            continue;
        }


        // start off by snagging the data we need.
        SecondaryData* info = &(goodImpactsLog[i]);
        BullyData* bully = &(initialImpactsLog[scratchpadGamma[info->tag]]);


        // write data to the structs
        finalImpactsLog[nSolutions + counter].bully.posBully[0] = bully->posBully[0];
        finalImpactsLog[nSolutions + counter].bully.posBully[1] = bully->posBully[1];
        finalImpactsLog[nSolutions + counter].bully.posBully[2] = bully->posBully[2];
        finalImpactsLog[nSolutions + counter].bully.angle = bully->angle;
        finalImpactsLog[nSolutions + counter].bully.velBully = bully->velBully;
        finalImpactsLog[nSolutions + counter].routeID = approach.solutionID;
        finalImpactsLog[nSolutions + counter].frames = info->frames;


        // then increment the counter
        counter++;
    }


    // and increase the total number of solutions we have logged.
    nSolutions += counter;
}

__global__ void fill_to_alpha(bool first, int size) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) {
        return;
    }

    // initialize with the identity function first off, and on successive rounds, you're copying scratchpad beta contents
    // into scratchpad alpha.
    if (first) {
        scratchpadAlpha[idx] = idx;
    }
    else {
        scratchpadAlpha[idx] = scratchpadBeta[idx];
    }
}




__global__ void add_to_gamma(int size, int samples) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) {
        return;
    }


    // spacing is the distance from each point in the sample set to the next. me is the position within scratchpad alpha
    // that is being considered. Assume you're not a duplicate until proven otherwise.
    int spacing = size / samples;
    int me = idx * spacing;
    bool duplicate = false;


    // iterate over stuff earlier in the sample set than you to see if you're a duplicate.
    // you're a duplicate if the bully speed and angle you're associated with match the bully speed and angle of something
    // earlier in the sample set than you.
    for (int i = 0; i < idx; i++) {
        if (initialImpactsLog[scratchpadAlpha[me]].velBully == initialImpactsLog[scratchpadAlpha[i * spacing]].velBully && initialImpactsLog[scratchpadAlpha[me]].angle == initialImpactsLog[scratchpadAlpha[i * spacing]].angle) {
            return;
        }
    }


    // fill in scratchpad gamma.
    int solIdx = atomicAdd(&nChecked, 1);
    if (solIdx > MAX_UNIQUE) {
        return;
    }
    scratchpadGamma[solIdx] = scratchpadAlpha[me];
}




__global__ void filter_to_beta(int alphasize, int gammasize) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= alphasize) {
        return;
    }


    // iterate over stuff in gamma to see whether you're a duplicate.
    for (int i = 0; i < gammasize; ++i) {
        if (initialImpactsLog[scratchpadAlpha[idx]].velBully == initialImpactsLog[scratchpadGamma[i]].velBully && initialImpactsLog[scratchpadAlpha[idx]].angle == initialImpactsLog[scratchpadGamma[i]].angle) {
            return;
        }
    }


    // fill in scratchpad beta.
    int solIdx = atomicAdd(&nUnchecked, 1);
    if (solIdx > alphasize) {
        return;
    }
    scratchpadBeta[solIdx] = scratchpadAlpha[idx];
}


// Function to extract specific columns (with help from ChatGPT).
int load_arrival_structs_from_csv(std::string csv_path, ApproachData* &arrival_data_array) {
    
    std::vector<std::unordered_map<std::string, std::string>> csv_data = parse_csv(csv_path);

    int nApproaches = csv_data.size();
    arrival_data_array = (struct ApproachData*)std::malloc(nApproaches * sizeof(struct ApproachData));

    int counter = 0;
    for (const auto& row : csv_data) {

        // Arrival Position X
        auto kv_pair = row.find(" Arrival Position X");
        if (kv_pair != row.end()) {
            arrival_data_array[counter].posArrive[0] = std::stof(kv_pair->second);
        }

        // Arrival Position Y
         kv_pair = row.find(" Arrival Position Y");
        if (kv_pair != row.end()) {
            arrival_data_array[counter].posArrive[1] = std::stof(kv_pair->second);
        }

        // Arrival Position Z
        kv_pair = row.find(" Arrival Position Z");
        if (kv_pair != row.end()) {
            arrival_data_array[counter].posArrive[2] = std::stof(kv_pair->second);
        }

        // Arrival Velocity
        kv_pair = row.find(" Arrival Velocity");
        if (kv_pair != row.end()) {
            arrival_data_array[counter].velArrive = std::stof(kv_pair->second);
        }

        // Arrival Angle
        kv_pair = row.find(" Arrival Angle");
        if (kv_pair != row.end()) {
            arrival_data_array[counter].facingArrive = std::stoi(kv_pair->second);
        }

        // Solution ID
        kv_pair = row.find(" solution ID");
        if (kv_pair != row.end()) {
            arrival_data_array[counter].solutionID = std::stoi(kv_pair->second);
        }

        counter++;
    }

    std::cout << "CSV Parsed.\n";

    return nApproaches;
}




int main(int argc, char* argv[]) {
    std::cout << "Starting Bully Impact Bruteforcer...\n";
    
    // initialize the prototype bully data. The X and Z are absolutely pinned down but the Y, angle, and velocity, may be messed with.
    struct BullyData bullyCentral;
    bullyCentral.posBully[0] = -3120.0f;
    bullyCentral.posBully[1] = -2976.0f;
    bullyCentral.posBully[2] = -896.0f;
    bullyCentral.angle = 11732;
    bullyCentral.velBully = 30000.0f;

    
    // initialize the float granularity for positions, and threads, and output file
    float granularity = 0.0078125f;
    int nThreads = 256;
    std::string outFile = "goodImpacts.csv";
    std::string inFile = "bloomVersion2.csv";
    bool verbose = false;

    
    // take input. Warning, I might have fucked up the file input reading.
    for (int i = 1; i < argc; i++) {
        if (!strcmp(argv[i], "-h") || !strcmp(argv[i], "--help")) {
            printf("BitFS Bully Impact Brute Forcer.\n");
            printf("This program accepts the following options:\n\n");
            printf("-an <angle>: Angle the bully has at time of impact. \n");
            printf("             Default: %d\n", bullyCentral.angle);
            printf("-vel <speed>: Velocity the bully has at time of impact. \n");
            printf("             Default: %f\n", bullyCentral.velBully);
            printf("-i: Path to the input file.\n");
            printf("    Default: %s\n", inFile.c_str());
            printf("-o: Path to the output file.\n");
            printf("    Default: %s\n", outFile.c_str());
            printf("-v: Verbose mode. Prints all parameters used in brute force.\n");
            printf("    Default: off\n");
            printf("-h --help: Prints this text.\n");
            exit(0);
        }
        else if (!strcmp(argv[i], "-an")) {
            bullyCentral.angle = std::stoi(argv[i + 1]);
            i += 1;
        }
        else if (!strcmp(argv[i], "-vel")) {
            bullyCentral.velBully = std::stoi(argv[i + 1]);
            i += 1;
        }
        else if (!strcmp(argv[i], "-o")) {
            outFile = argv[i + 1];
            i += 1;
        }
        else if (!strcmp(argv[i], "-i")) {
            inFile = argv[i + 1];
            i += 1;
        }
        else if (!strcmp(argv[i], "-v")) {
            verbose = true;
        }
        if (verbose) {
            printf("Bully Starting Angle: %d\n", bullyCentral.angle);
            printf("Bully Starting Speed: %f\n", bullyCentral.velBully);
        }
    }

    
    // load up our array of ways to arrive. Very high chance that something here is botched.
    struct ApproachData* arrivalList;
    int nApproaches = load_arrival_structs_from_csv(inFile, arrivalList);

    std::cout << "Cuda Mallocs Starting...\n";

    // initialize the memory for the scratchpads, unique initial impacts, good impacts (those which persist a long time)
    // and the final impact log.
    int* scratchpadAlphaGPU;
    hipMalloc((void**)&scratchpadAlphaGPU, MAX_INITIALS_PER_ARRIVAL * sizeof(int));
    int* scratchpadBetaGPU;
    hipMalloc((void**)&scratchpadBetaGPU, MAX_INITIALS_PER_ARRIVAL * sizeof(int));
    int* scratchpadGammaGPU;
    hipMalloc((void**)&scratchpadGammaGPU, MAX_UNIQUE * sizeof(int));
    BullyData* initialImpactsGPU;
    hipMalloc((void**)&initialImpactsGPU, MAX_INITIALS_PER_ARRIVAL * sizeof(BullyData));
    SecondaryData* goodImpactsGPU;
    hipMalloc((void**)&goodImpactsGPU, MAX_GOOD_IMPACTS_PER_ARRIVAL * sizeof(SecondaryData));
    ImpactData* finalImpactsGPU;
    hipMalloc((void**)&finalImpactsGPU, MAX_IMPACTS * sizeof(ImpactData));

    // BullyData* initialImpactsCPU = (BullyData*) std::malloc(sizeof(BullyData) * MAX_INITIALS_PER_ARRIVAL);

    
   // and get those pointers onto the GPU.
    copy_pointers_to_gpu << <1, 1 >> > (initialImpactsGPU, goodImpactsGPU, finalImpactsGPU, scratchpadAlphaGPU, scratchpadBetaGPU, scratchpadGammaGPU);
    
    // we'll be taking the overlap of the boxes "stuff within an Linfinity distance of 63 from Mario" and "stuff within an
    // Linfinity distance of 16 from the bully" to get our candidates for perturbed bully positions. Initialize a bunch of floats
    // pertaining to that. The first argument is 0 for min and 1 for max, and the second argument is 0 for x and 1 for z.
    float bpos[2][2];
    float mpos[2][2];
    float zone[2][2];
    bpos[0][0] = bullyCentral.posBully[0] - 16.0f;
    bpos[0][1] = bullyCentral.posBully[2] - 16.0f;
    bpos[1][0] = bullyCentral.posBully[0] + 16.0f;
    bpos[1][1] = bullyCentral.posBully[2] + 16.0f;

    std::cout << "Starting main loop...\n";

    // Start the clock
    auto computation_start = std::chrono::high_resolution_clock::now();

    // Now, iterate over possible ways that Mario could arrive near the bully.
    for (int i = 0; i < nApproaches; i++) {
        
        mpos[0][0] = arrivalList[i].posArrive[0] - 63.0f;
        mpos[0][1] = arrivalList[i].posArrive[2] - 63.0f;
        mpos[1][0] = arrivalList[i].posArrive[0] + 63.0f;
        mpos[1][1] = arrivalList[i].posArrive[2] + 63.0f;

        // compute intersection of boxes.
        zone[0][0] = fmaxf(bpos[0][0], mpos[0][0]);
        zone[0][1] = fmaxf(bpos[0][1], mpos[0][1]);
        zone[1][0] = fminf(bpos[1][0], mpos[1][0]);
        zone[1][1] = fminf(bpos[1][1], mpos[1][1]);
        
        // nx and nz are "if we take our rectangle of possible bully starting positions and discretize it according to the
        // float granularity we're likely to have, how many positions are there in the X and Z coordinates?". Maybe the
        // rectangles of viable bully positions don't intersect, in which case we'd expect some of the deltas to be negative
        // so we automatically move onto the next loop if that happens because no starting bully positions would work anyways.
        float deltax = zone[1][0] - zone[0][0];
        float deltaz = zone[1][1] - zone[0][1];
        int nx = (int)(deltax / granularity) + 1;
        int nz = (int)(deltaz / granularity) + 1;
        if (nx <= 0 || nz <= 0) {
            continue;
        }

        // we'll be testing nx * nz possible bully positions which *might* snap to the FST position after an impact.
        // so we set up the blocks, initialize to 0 solutions, push the solution counter to the GPU, test the bully positions
        // with the initial_assessment function, and pull the solution counter back out of the GPU. Bloom filtering to eliminate
        // duplicate solutions is done implicitly in the initial_assessment function.
        int nFirstBlocks = (nx * nz + nThreads - 1) / nThreads;
        int nInitialsCPU = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(nInitials), &nInitialsCPU, sizeof(int), 0, hipMemcpyHostToDevice);
        initial_assessment << <nFirstBlocks, nThreads >> > (arrivalList[i], bullyCentral, nx, nz, zone[0][0], zone[0][1], granularity);
        hipMemcpyFromSymbol(&nInitialsCPU, HIP_SYMBOL(nInitials), sizeof(int), 0, hipMemcpyDeviceToHost);

        
        // if no solutions or too many solutions, continue to the next loop or clip off the number of solutions.
        if (nInitialsCPU > MAX_INITIALS_PER_ARRIVAL) {
            fprintf(stderr, "Warning: The number of initial impacts has been exceeded. No more will be recorded. Increase the internal maximum to prevent this from happening.\n");
            nInitialsCPU = MAX_INITIALS_PER_ARRIVAL;
        }
        if (nInitialsCPU == 0) {
            continue;
        }

        std::cout << "Initial Solutions Found.\n";

        // and now for the black magic! We start off with everything unchecked and nothing checked. Make nChecked on the GPU be 0.
        int nUncheckedCPU = nInitialsCPU;
        int nCheckedCPU = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(nChecked), &nCheckedCPU, sizeof(int), 0, hipMemcpyHostToDevice);
        bool first = true;
        int blocksforAlpha = (nUncheckedCPU + nThreads - 1) / nThreads;
        fill_to_alpha << <blocksforAlpha, nThreads >> > (true, nUncheckedCPU);

        // While we still have stuff that isn't equivalent to something in our list of equivalence class representatives in
        // scratchpad Gamma... keep going.
        while (nUncheckedCPU > 0) {

            // copy the contents of beta over to alpha, or initialize alpha if it's the first loop. But if it's the first loop
            // then afterwards note that the first loop is done.
            int blocksforAlpha = (nUncheckedCPU + nThreads - 1) / nThreads;
            fill_to_alpha << <blocksforAlpha, nThreads >> > (first, nUncheckedCPU);
            if (first) {
                first = false;
            }

            // work out the number of points in our sample set. Basically, it starts off with the square root of the
            // number of unchecked points, but past that the sample size will tend to be the same as the number of
            // entries in scratchpad Gamma, doubling each time. Or if that's too big, it just tests everything that's left.
            // Then, nChecked (number of equivalence class representatives) is pulled out to the CPU so we know how big
            // our segment-of-interest on scratchpad Gamma is.
            int samples = max((int)sqrtf((float)nUncheckedCPU), min(nUncheckedCPU, nCheckedCPU));
            int blocksforGamma = (samples + nThreads - 1) / nThreads;
            add_to_gamma << <blocksforGamma, nThreads >> > (nUncheckedCPU, samples);
            hipMemcpyFromSymbol(&nCheckedCPU, HIP_SYMBOL(nChecked), sizeof(int), 0, hipMemcpyDeviceToHost);

            // initialize the number of uncheckeds to 0 on the GPU, filter out some stuff on scratchpad Alpha, and
            // then pull the number of uncheckeds (on beta) out so we can see how many unchecked entries are left. If 0, exit.
            int blocksforBeta = (nUncheckedCPU + nThreads - 1) / nThreads;
            int nBetaCPU = 0;
            hipMemcpyToSymbol(HIP_SYMBOL(nUnchecked), &nBetaCPU, sizeof(int), 0, hipMemcpyHostToDevice);
            filter_to_beta << <blocksforBeta, nThreads >> > (nUncheckedCPU, nCheckedCPU);
            hipMemcpyFromSymbol(&nBetaCPU, HIP_SYMBOL(nUnchecked), sizeof(int), 0, hipMemcpyDeviceToHost);
            nUncheckedCPU = nBetaCPU;
        }


        // we'll be testing (number of entries in scratchpad Gamma) bully positions which have promise, to see
        // whether they indeed snap to the FST position after impact, and how long they remain stable there. Same thing.
        // Set up the blocks, initialize to 0 solutions, push the solution counter to GPU, time-evolve the bully, and
        // pull the solution counter back out. Bloom filtering isn't done because there's negligible probability of having
        // two distinct impacts produce identical end states.
        int nSecondBlocks = (nCheckedCPU + nThreads - 1) / nThreads;
        int nImpactsCPU = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(nImpacts), &nImpactsCPU, sizeof(int), 0, hipMemcpyHostToDevice);
        time_evolution << <nSecondBlocks, nThreads >> > (nCheckedCPU);
        hipMemcpyFromSymbol(&nImpactsCPU, HIP_SYMBOL(nImpacts), sizeof(int), 0, hipMemcpyDeviceToHost);


        // again, clip if too many solutions and continue if no solutions.
        if (nImpactsCPU > MAX_GOOD_IMPACTS_PER_ARRIVAL) {
            fprintf(stderr, "Warning: The number of good impacts has been exceeded. No more will be recorded. Increase the internal maximum to prevent this from happening.\n");
            nImpactsCPU = MAX_GOOD_IMPACTS_PER_ARRIVAL;
        }
        if (nImpactsCPU == 0) {
            continue;
        }


        // append the data to the master log of solutions.
        append_info << <1, 1 >> > (arrivalList[i], nImpactsCPU);
    }

    // End the clock
    auto computation_end = std::chrono::high_resolution_clock::now();

    // Calculate the computation duration
    std::chrono::duration<double> duration = computation_end - computation_start;

    // Output the computation duration
    std::cout << "Computation Finished in: " << duration.count() << " seconds\n";

    // free up memory. Note that we don't free up our final log of solutions.
    hipFree(initialImpactsGPU);
    hipFree(goodImpactsGPU);
    hipFree(scratchpadAlphaGPU);
    hipFree(scratchpadBetaGPU);
    hipFree(scratchpadGammaGPU);


    // figure out how many solutions we have.

    int nSolutionsCPU = 0;
    hipMemcpyFromSymbol(&nSolutionsCPU, HIP_SYMBOL(nSolutions), sizeof(int), 0, hipMemcpyDeviceToHost);
    printf("%d solutions found!\n", nSolutionsCPU);

    printf("Copying Solutions to CPU...\n");
    // Start the clock
    auto copying_start = std::chrono::high_resolution_clock::now();

    // get the solutions from the GPU to the CPU.
    struct ImpactData* finalImpactLog = (struct ImpactData*)std::malloc(nSolutionsCPU * sizeof(struct ImpactData));
    hipMemcpy(finalImpactLog, finalImpactsGPU, nSolutionsCPU * sizeof(struct ImpactData), hipMemcpyDeviceToHost);

    // End the clock
    auto copying_end = std::chrono::high_resolution_clock::now();

    // Calculate the copying duration
    duration = copying_end - copying_start;

    // Output the copying duration
    std::cout << "Copying Finished in: " << duration.count() << " seconds\n";

    std::cout << "Writing to file...\n";
    // Start the clock
    auto writing_start = std::chrono::high_resolution_clock::now();

    // ok, at this point all our GPU shit is over and we've got a bunch of solutions in a table. It's time
    // to start writing this shit into a file.
    std::ofstream wf(outFile);
    wf << std::fixed;
    wf << "Bully Position X, Bully Position Z, ";
    wf << "Bully Angle, Bully Velocity, ";
    wf << "solution ID, Frames of Stability" << std::endl;
    for (int m = 0; m < nSolutionsCPU; m++) {
        wf << finalImpactLog[m].bully.posBully[0] << ", " << finalImpactLog[m].bully.posBully[2] << ", ";
        wf << finalImpactLog[m].bully.angle << ", " << finalImpactLog[m].bully.velBully << ", ";
        wf << finalImpactLog[m].routeID << ", " << finalImpactLog[m].frames << std::endl;
    }
    wf.close();

    // End the clock
    auto writing_end = std::chrono::high_resolution_clock::now();

    // Calculate the writing duration
    duration = writing_end - writing_start;

    // Output the writing duration
    std::cout << "Writing Finished in: " << duration.count() << " seconds\n";

    // free up memory.
    std::free(arrivalList);
    std::free(finalImpactLog);
    hipFree(finalImpactsGPU);


    // end
    printf("Complete!");
    return 0;
}